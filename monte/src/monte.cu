// includes, system

#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <time.h>

// includes CUDA
//#include <helper_cuda.h>  //needed for findCudaDevice()
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

/******************************** DEVICE CODE ********************************/

__global__ void init_random_blocks(int seed, hiprandState_t *state)  {
    hiprand_init(seed, blockIdx.x, 0, &state[blockIdx.x]);
}

__global__ void monteBlocks(hiprandState_t *states, int *throws, int *hits)  {
	double x, y;
	x = hiprand_uniform_double(&states[blockIdx.x]);
	y = hiprand_uniform_double(&states[blockIdx.x]);
	throws[blockIdx.x]++;
	if (sqrt(x*x + y*y) <= 1.)  {
		hits[blockIdx.x]++;
	}
}

__global__ void init_random_threads(int seed, hiprandState_t *state)  {
    hiprand_init(seed, threadIdx.x, 0, &state[threadIdx.x]);
}

__global__ void monteThreads(hiprandState_t *states, int *throws, int *hits)  {
	double x, y;
	x = hiprand_uniform_double(&states[threadIdx.x]);
	y = hiprand_uniform_double(&states[threadIdx.x]);
	throws[threadIdx.x]++;
	if (sqrt(x*x + y*y) <= 1.)  {
		hits[threadIdx.x]++;
	}
}

__global__ void init_random_threads_blocks(int seed, hiprandState_t *state)  {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, index, 0, &state[index]);
}

__global__ void init_monteThreadsBlocks(int *throws, int *hits)  {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	hits[index] = throws[index] = 0;
}

__global__ void monteThreadsBlocks(int n, hiprandState_t *states, int *throws, int *hits)  {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	double x, y;
	/*
	x = curand_uniform_double(&states[index]);
	y = curand_uniform_double(&states[index]);
	throws[index]++;
	if (sqrt(x*x + y*y) <= 1.)  {
		hits[index]++;
	}
	*/

	for (int i = index; i < n; i += stride)  {
		x = hiprand_uniform_double(&states[i]);
		y = hiprand_uniform_double(&states[i]);
		throws[i]++;
		if (sqrt(x*x + y*y) <= 1.)  {
			hits[i]++;
		}
	}

}

/********************************* HOST CODE *********************************/

void pi(int argc, char **argv)
{
    printf("%s Starting...\n\n", argv[0]);
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //print devID = findCudaDevice(argc, (const char **)argv);


    int n = 128 * 128 * 256;
    hiprandState_t *state;
    int state_size = n * sizeof(hiprandState_t);
    hipMallocManaged(&state, state_size);

    int blockSize = 256;
    int numBlocks = (n + blockSize -1) / blockSize;

    unsigned int t = time(0);
    //t = 1234;
//    init_random_blocks<<<n,1>>>(t, state);
//    init_random_threads<<<numBlocks,blockSize>>>(t, state);
    init_random_threads_blocks<<<numBlocks,blockSize>>>(t, state);
    hipDeviceSynchronize();

    int size = n * sizeof(int);
    int *hits;
    hipMallocManaged(&hits, size);
    int *throws;
    hipMallocManaged(&throws, size);

    init_monteThreadsBlocks<<<numBlocks,blockSize>>>(throws, hits);
    hipDeviceSynchronize();

//    monteBlocks<<<n,1>>>(state, throws, hits);
//    monteThreads<<<1,n>>>(state, throws, hits);
    monteThreadsBlocks<<<numBlocks,blockSize>>>(n, state, throws, hits);
    hipDeviceSynchronize();

    int total_hits = 0;
    int total_throws = 0;
    for (int i=0; i<n; i++)  {
    	total_hits += hits[i];
    	total_throws += throws[i];
    }

    double pie = 4. * double(total_hits) / double(total_throws);
    std::cout << pie << "    " << total_throws << "\n";

    hipFree(state);
    hipFree(hits);
    hipFree(throws);
}

// Program main
int main(int argc, char **argv) {
    pi(argc, argv);
}

// some hints here
// http://stackoverflow.com/questions/11832202/cuda-random-number-generating
//
// this is pretty good...look at multi core implementation at bottom
// http://cs.umw.edu/~finlayson/class/fall16/cpsc425/notes/cuda-random.html
//
// good article
// http://stackoverflow.com/questions/26650391/generate-random-number-within-a-function-with-curand-without-preallocation



